#include "hip/hip_runtime.h"

#include "MurmurHash2.cuh"

__device__ __forceinline__
uint64_t MurmurHash64A_GPU(const void* key, int len, uint64_t seed = 0x1e35a7bdUL)
{
    return MurmurHash64A(key, len, seed);
}

//d_in: 원본 바이트 버퍼, d_out: 해쉬 결과 배열(device)
//blk_sz, N: 블락 사이즈, 개수
__global__ void hash_kernel(const char* d_in, uint64_t* d_out, size_t blk_sz, size_t N)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        d_out[i] = MurmurHash64A_GPU(d_in + i*blk_sz, blk_sz);
}